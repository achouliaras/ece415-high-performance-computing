#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "gputimer.h"
unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005 

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }
////////////////////////////////////////////////////////////////////////////////
// GPU row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convRowGPU( float *d_Dst, float *d_Src, float *d_Filter, 
                       int imageW, int imageH, int filterR ){
    int ix= blockDim.x * blockIdx.x + threadIdx.x +filterR;
    int iy= blockDim.y * blockIdx.y + threadIdx.y +filterR;
    int ik;
    float sum=0;
    for (ik = -filterR; ik <= filterR; ik++) {
        int d = ix + ik;

        sum += d_Src[iy * (imageW+ 2*filterR) + d] * d_Filter[filterR - ik];    
    }
    d_Dst[iy * (imageW+ 2*filterR) + ix] = sum;
}
////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}
////////////////////////////////////////////////////////////////////////////////
// GPU column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convColGPU( float *d_Dst, float *d_Src, float *d_Filter, 
                       int imageW, int imageH, int filterR ){
    int ix= blockDim.x * blockIdx.x + threadIdx.x +filterR;
    int iy= blockDim.y * blockIdx.y + threadIdx.y +filterR;
    int ik;
    float sum = 0;

      for (ik = -filterR; ik <= filterR; ik++) {
        int d = iy + ik;

        sum += d_Src[d * (imageW+ 2*filterR) + ix] * d_Filter[filterR - ik];   
        
      }
      d_Dst[iy * (imageW+ 2*filterR) + ix] = sum;
}

////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;
    
    float
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU;
    
    unsigned int imageW;
    unsigned int imageH;
    unsigned int i;
    int rval;
    
    struct timespec  tv1, tv2;

	printf("Enter filter radius : ");
	rval=scanf(" %d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    rval=scanf(" %d", &imageW);
    imageH = imageW;
    unsigned int block_size=(imageW<=32) ? imageW : 32;
    unsigned int gridsize=(imageW)/block_size;
    if (imageW % block_size != 0)
        gridsize++;
    dim3 dimGrid( gridsize, gridsize);
    dim3 dimBlock( block_size, block_size);   // 64x64 den douleuei
    
    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));
    
    // if any memory allocation failed, report an error message
    if(h_Filter == 0 || h_Input == 0 || h_Buffer == 0 || h_OutputCPU == 0 || h_OutputGPU == 0){
        printf("CPU couldn't allocate memory\n");
        return 1;
    }
    
    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }
    
    // GPU code section starts here
    GpuTimer timer;
    //Memory Allocation
    hipMalloc((void**)&d_Filter   , FILTER_LENGTH * sizeof(float));
    hipMalloc((void**)&d_Input    , (imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(float));
    hipMalloc((void**)&d_Buffer   , (imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(float));
    hipMalloc((void**)&d_OutputGPU, (imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(float));
    
    // if any memory allocation failed, report an error message
    if(d_Filter == 0 || d_Input == 0 || d_Buffer == 0 || d_OutputGPU == 0 ){
        printf("GPU couldn't allocate memory\n");
        return 1;
    }
    //Memory Initialization
    hipMemset(d_Filter   ,0, FILTER_LENGTH * sizeof(float));
    hipMemset(d_Input    ,0, (imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(float));
    hipMemset(d_Buffer   ,0, (imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(float));
    hipMemset(d_OutputGPU,0, (imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(float));
    
    printf("GPU computation...\n");
    //timer.Start();                       // START GPU TIMER
    
    hipMemcpy(d_Filter,h_Filter, FILTER_LENGTH * sizeof(float)  , hipMemcpyHostToDevice);
    for(i=0;i<imageH;i++){
        hipMemcpy(d_Input+((i+filter_radius)*(imageW+2*filter_radius)+filter_radius) ,h_Input+(i*imageW) ,\
                                                        imageW * sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(d_Buffer+((i+filter_radius)*(imageW+2*filter_radius)+filter_radius) ,h_Buffer+(i*imageW),\
                                                        imageW * sizeof(float), hipMemcpyHostToDevice);
    }
    
    timer.Start();                       // START GPU TIMER
    // GPU convolution kata grammes
    convRowGPU<<<dimGrid, dimBlock>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius); 
    // convolution kata sthles
    convColGPU<<<dimGrid, dimBlock>>>(d_OutputGPU, d_Buffer, d_Filter, imageW, imageH, filter_radius); 
    timer.Stop();                        // FINISH GPU TIMER
    
    
    for(i=0;i<imageH;i++){
        hipMemcpy(h_OutputGPU+(i*imageW), d_OutputGPU+((i+filter_radius)*(imageW+2*filter_radius)+filter_radius) ,\
                                                        imageW * sizeof(float), hipMemcpyDeviceToHost);
     }
    //timer.Stop();                        // FINISH GPU TIMER
    
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);
    
    //convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    //convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);
    
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  
    
    //Standard accuracy check
    rval = 0;
    for (i = 0; i < imageW * imageH; i++) {
        if( ABS(h_OutputGPU[i]-h_OutputCPU[i]) > accuracy){
            printf("The two images differ...\n");
            printf("%d ->%f \t ->%f\n",i,h_OutputGPU[i],h_OutputCPU[i]);
            rval = 1;
            break;
        }
    }
    if(rval == 0){
        printf("The two images are the SAME!!!\n");
    }
    
    // find max accuracy for given filter radius
    /*
    double acc=0;
    for( acc=0;acc<10; acc=acc+0.000001){
        rval = 0;
        for (i = 0; i < imageW * imageH; i++) {
            if( ABS(h_OutputGPU[i]-h_OutputCPU[i]) > acc){
                //printf("The two images differ...%lf\n",acc);
                rval = 1;
                break;
            }
        }
        if(rval == 0){
            printf("The two images are the SAME!!! %lf\n",acc);
            break;
        }
    }
    */
    
    printf("GPU Time elapsed = %-10g s\n", timer.Elapsed()/1000.0);
    printf("CPU time elapsed = %-10g s\n",
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec));
    
    // free all the allocated memory
    hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputGPU);
    free(h_OutputGPU);
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    
    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceSynchronize();
    cudaCheckError();
    hipDeviceReset();
    
    return 0;
}
