#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "gputimer.h"
unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

//#define CPU_compute

__device__ __constant__ double d_Filter[2*256+1]={0};

////////////////////////////////////////////////////////////////////////////////
// GPU row convolution filter
////////////////////////////////////////////////////////////////////////////////

__global__ void convRowGPU( double *d_Dst, double *d_Src,
                       int imageW, int imageH, int filterR ){
    int ix= blockDim.x * blockIdx.x + threadIdx.x +filterR;
    int iy= blockDim.y * blockIdx.y + threadIdx.y +filterR;
    int ik;
    double sum=0;
    for (ik = -filterR; ik <= filterR; ik++) {
        int d = ix + ik;

        sum += d_Src[iy * (imageW+ 2*filterR) + d] * d_Filter[filterR - ik];
    }
    d_Dst[iy * (imageW+ 2*filterR) + ix] = sum;
}
////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter,
                       int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}
////////////////////////////////////////////////////////////////////////////////
// GPU column convolution filter
////////////////////////////////////////////////////////////////////////////////

__global__ void convColGPU( double *d_Dst, double *d_Src,
                       int imageW, int imageH, int filterR ){
    int ix= blockDim.x * blockIdx.x + threadIdx.x +filterR;
    int iy= blockDim.y * blockIdx.y + threadIdx.y +filterR;
    int ik;
    double sum = 0;
    for (ik = -filterR; ik <= filterR; ik++) {
      int d = iy + ik;

      sum += d_Src[d * (imageW+ 2*filterR) + ix] * d_Filter[filterR - ik];
    }
    d_Dst[iy * (imageW+ 2*filterR) + ix] = sum;
}
////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {

    double
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;

    double
    //*d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU;

    unsigned int imageW;
    unsigned int imageH;
    unsigned int i;
    int rval;

    #ifdef CPU_compute
    struct timespec  tv1, tv2;
    #endif

	printf("Enter filter radius : ");
	rval=scanf(" %d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    rval=scanf(" %d", &imageW);
    if (rval== -1){
        printf("ERROR scanf");
    }
    imageH = imageW;
    unsigned int block_size=(imageW<=32) ? imageW : 32;
    unsigned int gridsize=(imageW)/block_size;
    if (imageW % block_size != 0)
        gridsize++;
    dim3 dimGrid( gridsize, gridsize);
    dim3 dimBlock( block_size, block_size);   // 64x64 den douleuei

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    h_Input     = (double *)malloc(imageW * imageH * sizeof(double));
    h_Buffer    = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputGPU = (double *)malloc(imageW * imageH * sizeof(double));

    // if any memory allocation failed, report an error message
    if(h_Filter == 0 || h_Input == 0 || h_Buffer == 0 || h_OutputCPU == 0 || h_OutputGPU == 0){
        printf("CPU couldn't allocate memory\n");
        return 1;
    }

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (double)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
    }

    // GPU code section starts here
    GpuTimer timer;
    //Memory Allocation
    //hipMalloc((void**)&d_Filter   , FILTER_LENGTH * sizeof(double));
    hipMalloc((void**)&d_Input    , (imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(double));
    hipMalloc((void**)&d_Buffer   , (imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(double));
    hipMalloc((void**)&d_OutputGPU, (imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(double));

    // if any memory allocation failed, report an error message
    //d_Filter == 0 ||
    if( d_Input == 0 || d_Buffer == 0 || d_OutputGPU == 0 ){
        printf("GPU couldn't allocate memory\n");
        return 1;
    }
    //Memory Initialization
    //hipMemset(d_Filter   ,0, FILTER_LENGTH * sizeof(double));
    hipMemset(d_Input    ,0, (imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(double));
    hipMemset(d_Buffer   ,0, (imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(double));
    hipMemset(d_OutputGPU,0, (imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(double));

    printf("GPU computation...\n");
    timer.Start();                       // START GPU TIMER

    hipMemcpyToSymbol(HIP_SYMBOL(d_Filter),h_Filter, FILTER_LENGTH * sizeof(double), 0 , hipMemcpyHostToDevice);
    for(i=0;i<imageH;i++){
        hipMemcpy(d_Input+((i+filter_radius)*(imageW+2*filter_radius)+filter_radius) ,h_Input+(i*imageW) ,\
                                                        imageW * sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(d_Buffer+((i+filter_radius)*(imageW+2*filter_radius)+filter_radius) ,h_Buffer+(i*imageW),\
                                                        imageW * sizeof(double), hipMemcpyHostToDevice);
    }

    //timer.Start();                       // START GPU TIMER
    // GPU convolution kata grammes
    convRowGPU<<<dimGrid, dimBlock>>>(d_Buffer, d_Input, imageW, imageH, filter_radius);
    // convolution kata sthles
    convColGPU<<<dimGrid, dimBlock>>>(d_OutputGPU, d_Buffer, imageW, imageH, filter_radius);
    //timer.Stop();                        // FINISH GPU TIMER

    for(i=0;i<imageH;i++){
        hipMemcpy(h_OutputGPU+(i*imageW), d_OutputGPU+((i+filter_radius)*(imageW+2*filter_radius)+filter_radius) ,\
                                                        imageW * sizeof(double), hipMemcpyDeviceToHost);
     }
    timer.Stop();                        // FINISH GPU TIMER

    #ifdef CPU_compute

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas

    //Standard accuracy check
    rval = 0;
    for (i = 0; i < imageW * imageH; i++) {
        if( ABS(h_OutputGPU[i]-h_OutputCPU[i]) > accuracy){
            printf("The two images differ...\n");
            printf("%d ->%f \t ->%f\n",i,h_OutputGPU[i],h_OutputCPU[i]);
            rval = 1;
            break;
        }
    }
    if(rval == 0){
        printf("The two images are the SAME!!!\n");
    }

    // find max accuracy for given filter radius
    /*
    double acc=0;
    for( acc=0;acc<10; acc=acc+0.000001){
        rval = 0;
        for (i = 0; i < imageW * imageH; i++) {
            if( ABS(h_OutputGPU[i]-h_OutputCPU[i]) > acc){
                //printf("The two images differ...%lf\n",acc);
                rval = 1;
                break;
            }
        }
        if(rval == 0){
            printf("The two images are the SAME!!! %lf\n",acc);
            break;
        }
    }
    */
    #endif

    printf("GPU Time elapsed = %-10g s\n", timer.Elapsed()/1000.0);
    #ifdef CPU_compute
    printf("CPU time elapsed = %-10g s\n",
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec));
    #endif
    // free all the allocated memory
    //hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputGPU);
    free(h_OutputGPU);
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceSynchronize();
    cudaCheckError();
    hipDeviceReset();

    return 0;
}
