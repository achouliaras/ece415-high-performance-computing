#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "gputimer.h"
unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005

#define TILE_SIZE 256

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

//#define CPU_compute

__device__ __constant__ double d_Filter[2*256+1]={0};

////////////////////////////////////////////////////////////////////////////////
// GPU row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convRowGPU( double *d_Dst, double *d_Src,
                       int tileW, int imageW, int imageH, int filterR ){
    int tx=threadIdx.x +filterR , ty=threadIdx.y;
    int ix= blockDim.x * blockIdx.x + tx ;
    int iy= blockDim.y * blockIdx.y + ty ;
    int ik,d;
    double sum=0;
    extern __shared__ double s_src[]; // size (block_size + 2*filter_radius)*block_size*sizeof(double)
    for (ik = -filterR; ik <= filterR; ik ++) {
      d = ix + ik;
      s_src[ty * (blockDim.x + 2*filterR) + tx + ik]=d_Src[iy * (tileW+ 2*filterR) + d];
    }
    __syncthreads();
    for (ik = -filterR; ik <= filterR; ik++) {
        d = tx + ik;

        sum += s_src[ty * (blockDim.x + 2*filterR) +d] * d_Filter[filterR - ik];
    }
    d_Dst[iy * (tileW+ 2*filterR) + ix] = sum;
}
////////////////////////////////////////////////////////////////////////////////
// GPU column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convColGPU( double *d_Dst, double *d_Src,
                       int tileW, int imageW, int imageH, int filterR ){
    int tx=threadIdx.x , ty=threadIdx.y +filterR;
    int ix= blockDim.x * blockIdx.x + tx ;
    int iy= blockDim.y * blockIdx.y + ty ;
    int ik,d;
    double sum = 0;
    extern __shared__ double s_src[]; // size (block_size + 2*filter_radius)*block_size*sizeof(double)
    for (ik = -filterR; ik <= filterR; ik ++) {
      d = iy + ik;
      s_src[(ty + ik) * (blockDim.y) + tx]=d_Src[d * (tileW) + ix];
    }
    __syncthreads();
    for (ik = -filterR; ik <= filterR; ik++) {
      d = ty + ik;

      sum += s_src[d * (blockDim.y) + tx] * d_Filter[filterR - ik];
    }
    d_Dst[iy * (tileW) + ix] = sum;
}
////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter,
                       int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}
////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {

    double *h_Filter, *h_Input, *h_Buffer, *h_OutputCPU, *h_OutputGPU, *h_input_pad, *h_input_pad0;
    //double *d_Filter;
    double *d_Input, *d_Buffer, *d_OutputGPU;
    unsigned int imageW, imageH, tileW, i,j,k,l;
    int rval;

    #ifdef CPU_compute
    struct timespec  tv1, tv2;
    #endif
	  printf("Enter filter radius : ");
	  rval=scanf(" %d", &filter_radius);
    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    rval=scanf(" %d", &imageW);
    if (rval== -1){
        printf("ERROR scanf");
    }
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    h_Input     = (double *)malloc(imageW * imageH * sizeof(double));
    h_Buffer    = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputGPU = (double *)malloc(imageW * imageH * sizeof(double));
    h_input_pad = (double *)malloc((imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(double));
    h_input_pad0 = (double *)malloc((imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(double));
    // if any memory allocation failed, report an error message
    if(h_Filter == 0 || h_Input == 0 || h_Buffer == 0 || h_OutputCPU == 0 || h_OutputGPU == 0 || h_input_pad == 0){
        printf("CPU couldn't allocate memory\n");
        return 1;
    }
    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.
    srand(200);
    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (double)(rand() % 16);
    }
    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
    }

    ////// GPU code section starts here //////
    unsigned int block_size=(imageW<=32) ? imageW : 32;
    if (imageW>TILE_SIZE) {
      tileW=TILE_SIZE;
    }
    else {
      tileW=imageW;
    }
    unsigned int gridsize=(tileW)/block_size;
    if (imageW % block_size != 0)
      gridsize++;
    dim3 dimGrid( gridsize, gridsize);
    dim3 dimBlock( block_size, block_size);   // 32x32
    GpuTimer timer;
    //Memory Allocation
    //hipMalloc((void**)&d_Filter   , FILTER_LENGTH * sizeof(double));
    hipMalloc((void**)&d_Input    , (tileW + 2*filter_radius)* tileW * sizeof(double));
    hipMalloc((void**)&d_Buffer   , (tileW + 2*filter_radius)* tileW * sizeof(double));
    hipMalloc((void**)&d_OutputGPU, (tileW + 2*filter_radius)* tileW * sizeof(double));
    // if any memory allocation failed, report an error message
    if(d_Filter == 0 || d_Input == 0 || d_Buffer == 0 || d_OutputGPU == 0 ){
        printf("GPU couldn't allocate memory\n");
        return 1;
    }
    //Memory Initialization
    //hipMemset(d_Filter   ,0, FILTER_LENGTH * sizeof(double));
    hipMemset(d_Input    ,0, (tileW + 2*filter_radius)* tileW * sizeof(double));
    hipMemset(d_Buffer   ,0, (tileW + 2*filter_radius)* tileW * sizeof(double));
    hipMemset(d_OutputGPU,0, (tileW + 2*filter_radius)* tileW * sizeof(double));
    memset(h_input_pad,0, (imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(double));
    memset(h_input_pad0,0, (imageW + 2*filter_radius)* (imageH + 2*filter_radius) * sizeof(double));
    printf("GPU computation...\n");
    //timer.Start();                       // START GPU TIMER

    hipMemcpyToSymbol(HIP_SYMBOL(d_Filter),h_Filter, FILTER_LENGTH * sizeof(double),0  , hipMemcpyHostToDevice);
    int shared_size = (block_size + 2*filter_radius)*block_size*sizeof(double);
    for(i=0;i<imageW;i++){
        memcpy(h_input_pad0+((i+filter_radius)*(imageW+2*filter_radius)+filter_radius),\
               h_Input+(i*imageW),\
               imageW * sizeof(double));
    }
    timer.Start();                       // START GPU TIMER
    for (l=filter_radius; l < (imageH+filter_radius); l+=tileW) {
      for (k=0; k < (imageW-filter_radius); k+=tileW) {
        //Move data from Host to GPU
        for (j=l; j< l + tileW; j++) {
            hipMemcpy(d_Input + (j-l)*(tileW + 2*filter_radius),\
                       h_input_pad0+j*(imageW + 2*filter_radius)+k,\
                      (tileW + 2*filter_radius) * sizeof(double),hipMemcpyHostToDevice);
        }
        // GPU convolution kata grammes
        convRowGPU<<<dimGrid, dimBlock, shared_size>>>(d_Buffer, d_Input, tileW, imageW, imageH, filter_radius);
        // Move data from GPU back to Host
        for (j=l; j< l + tileW; j++) {
            hipMemcpy(h_input_pad + j*(imageW + 2*filter_radius)+k+filter_radius,\
                       d_Buffer + (j-l)*(tileW + 2*filter_radius)+filter_radius,\
                       (tileW) * sizeof(double),hipMemcpyDeviceToHost);
        }
      }
    }
    cudaCheckError();
    for (l=0; l < (imageH-filter_radius); l+=tileW) {
      for (k=filter_radius; k < (imageW+filter_radius); k+=tileW) {
        //Move data from Host to GPU
        for (j=l; j< l + tileW+2*filter_radius; j++) {
            hipMemcpy(d_Buffer + (j-l)*tileW, h_input_pad+(j*(imageH + 2*filter_radius))+k,\
                      tileW * sizeof(double), hipMemcpyHostToDevice);
        }
        // GPU convolution kata sthles
        convColGPU<<<dimGrid, dimBlock, shared_size>>>(d_OutputGPU, d_Buffer, tileW, imageW, imageH, filter_radius);
        // Move data from GPU back to Host
        for (j=l+filter_radius; j< l + tileW+ filter_radius; j++) {
          hipMemcpy(h_input_pad0+(j)*(imageH+ 2*filter_radius)+k, d_OutputGPU+(j-l)*tileW,\
                    tileW*sizeof(double), hipMemcpyDeviceToHost);
        }
      }
    }
    timer.Stop();                        // FINISH GPU TIMER

    for(i=0;i<imageW;i++){
        memcpy(h_OutputGPU+(i*imageW),\
               h_input_pad0+((i+filter_radius)*(imageW+2*filter_radius)+filter_radius),\
               imageW * sizeof(double));
    }

    #ifdef CPU_compute
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas
    //Standard accuracy check
    rval = 0;
    for (i = 0; i < imageW * imageH; i++) {
        if( ABS(h_OutputGPU[i]-h_OutputCPU[i]) > accuracy){
            printf("The two images differ...\n");
            printf("%d ->%f \t ->%f\n",i,h_OutputGPU[i],h_OutputCPU[i]);
            rval = 1;
            break;
        }
    }
    if(rval == 0){
        printf("The two images are the SAME!!!\n");
    }
    //printf("%d ->%f \t ->%f\n",i,h_OutputGPU[i],h_OutputCPU[i]);
    // find max accuracy for given filter radius
    /*
    double acc=0;
    for( acc=0;acc<10; acc=acc+0.000001){
        rval = 0;
        for (i = 0; i < imageW * imageH; i++) {
            if( ABS(h_OutputGPU[i]-h_OutputCPU[i]) > acc){
                //printf("The two images differ...%lf\n",acc);
                rval = 1;
                break;
            }
        }
        if(rval == 0){
            printf("The two images are the SAME!!! %lf\n",acc);
            break;
        }
    }
    */
    #endif
    printf("GPU Time elapsed = %-10g s\n", timer.Elapsed()/1000.0);
    #ifdef CPU_compute
    printf("CPU time elapsed = %-10g s\n",
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec));
    #endif
    // free all the allocated memory
    //hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputGPU);
    free(h_OutputGPU);
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceSynchronize();
    cudaCheckError();
    hipDeviceReset();
    return 0;
}
