/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//#include "gputimer.h"
unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }
////////////////////////////////////////////////////////////////////////////////
//GPU row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convRowGPU( float *d_Dst, float *d_Src, float *d_Filter, 
                       int imageW, int imageH, int filterR ){
    int ix=  threadIdx.x;
    int iy=  threadIdx.y;
    int ik;
    float sum=0;
    for (ik = -filterR; ik <= filterR; ik++) {
        int d = ix + ik;

        if (d >= 0 && d < imageW) {
          sum += d_Src[iy * imageW + d] * d_Filter[filterR - ik];
        }     
    }
    d_Dst[iy * imageW + ix] = sum;
}
////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}
////////////////////////////////////////////////////////////////////////////////
//GPU column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convColGPU( float *d_Dst, float *d_Src, float *d_Filter, 
                       int imageW, int imageH, int filterR ){
    int ix=  threadIdx.x;
    int iy=  threadIdx.y;
    int ik;
    float sum = 0;

      for (ik = -filterR; ik <= filterR; ik++) {
        int d = iy + ik;

        if (d >= 0 && d < imageH) {
          sum += d_Src[d * imageW + ix] * d_Filter[filterR - ik];
        }   
      }
      d_Dst[iy * imageW + ix] = sum;
}

////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;
    
    float
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU;
    
    unsigned int imageW;
    unsigned int imageH;
    unsigned int i;
    int rval;

	printf("Enter filter radius : ");
	rval=scanf(" %d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    rval=scanf(" %d", &imageW);
    imageH = imageW;

    dim3 dimGrid(1, 1);
    dim3 dimBlock(imageW,imageH);   // 64x64 den douleuei
    
    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));
    
    // if any memory allocation failed, report an error message
    if(h_Filter == 0 || h_Input == 0 || h_Buffer == 0 || h_OutputCPU == 0 || h_OutputGPU == 0){
        printf("CPU couldn't allocate memory\n");
        return 1;
    }
    
    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }
    
    
    // GPU code section starts here
    
    //Memory Allocation
    hipMalloc((void**)&d_Filter   , FILTER_LENGTH * sizeof(float));
    hipMalloc((void**)&d_Input    , imageW * imageH * sizeof(float));
    hipMalloc((void**)&d_Buffer   , imageW * imageH * sizeof(float));
    hipMalloc((void**)&d_OutputGPU, imageW * imageH * sizeof(float));
    
    // if any memory allocation failed, report an error message
    if(d_Filter == 0 || d_Input == 0 || d_Buffer == 0 || d_OutputGPU == 0 ){
        printf("GPU couldn't allocate memory\n");
        return 1;
    }

    hipMemcpy(d_Filter,h_Filter, FILTER_LENGTH * sizeof(float)  , hipMemcpyHostToDevice);
    hipMemcpy(d_Input ,h_Input , imageW * imageH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Buffer,h_Buffer, imageW * imageH * sizeof(float), hipMemcpyHostToDevice);
    
    printf("GPU computation...\n");
    // GPU convolution kata grammes
    convRowGPU<<<dimGrid, dimBlock>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius); 
    // convolution kata sthles
    convColGPU<<<dimGrid, dimBlock>>>(d_OutputGPU, d_Buffer, d_Filter, imageW, imageH, filter_radius); 
    
   
    hipMemcpy(h_OutputGPU, d_OutputGPU , imageW * imageH * sizeof(float), hipMemcpyDeviceToHost);
    
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles


    // Kanete th sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  
    
    // find max accuracy for given filter radius
    /*
    double acc=0;
    for( acc=0;acc<10; acc=acc+0.000001){
        rval = 0;
        for (i = 0; i < imageW * imageH; i++) {
            if( ABS(h_OutputGPU[i]-h_OutputCPU[i]) > acc){
                //printf("The two images differ...%lf\n",acc);
                rval = 1;
                break;
            }
        }
        if(rval == 0){
            printf("The two images are the SAME!!! %lf\n",acc);
            break;
        }
    }
    */
    // Standard accuracy check
    rval = 0;
    for (i = 0; i < imageW * imageH; i++) {
        if( ABS(h_OutputGPU[i]-h_OutputCPU[i]) > accuracy){
            printf("The two images differ...\n");
            rval = 1;
            break;
        }
    }
    if(rval == 0){
        printf("The two images are the SAME!!!\n");
    }
    
    // free all the allocated memory
    hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputGPU);
    free(h_OutputGPU);
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    
    hipDeviceSynchronize();
    cudaCheckError();
    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceSynchronize();
    cudaCheckError();
    hipDeviceReset();


    return 0;
}
